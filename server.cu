#include "hip/hip_runtime.h"
#include <signal.h>
#include <dlfcn.h>
#include <iostream>
#include <string>
#include <thread>
#include <chrono>
#include <atomic>
#include <stdlib.h>
#include <fstream>
#include <sys/mman.h>
#include <sys/stat.h>        /* For mode constants */
#include <fcntl.h>           /* For O_* constants */
#include <unistd.h>
#include <cassert>
#include <functional>
#include <unordered_map>
#include <cxxabi.h>
#include <map>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "libipc/ipc.h"

#include "tally/util.h"
#include "tally/def.h"

__global__ void vectorAddKernel(const float* A, const float* B, float* C, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        C[i] = A[i] + B[i];
    }
}

int magic;
int version;
unsigned long long* fatbin_data;
uint32_t fatBinSize;

std::atomic<bool> is_quit__ {false};
ipc::channel *send_ipc = nullptr;
ipc::channel *recv_ipc = nullptr;
std::map<void *, std::vector<uint32_t>> _kernel_addr_to_args;
std::map<std::string, void *> _kernel_name_to_addr;
std::map<void *, void *> _kernel_client_addr_mapping;
std::vector<std::pair<void *, std::string>> register_queue;

void handle_cudaMalloc(struct cudaMallocArg *arg)
{
    static hipError_t (*lcudaMalloc) (void **, size_t );
	if (!lcudaMalloc) {
		lcudaMalloc = (hipError_t (*) (void **, size_t )) dlsym(RTLD_NEXT, "hipMalloc");
	}
	assert(lcudaMalloc);

    void *devPtr;
    hipError_t err = hipMalloc(&devPtr, arg->size);

    struct cudaMallocResponse res { devPtr, err };
    while(!send_ipc->send((void *) &res, sizeof(struct cudaMallocResponse))) {
        send_ipc->wait_for_recv(1);
    }
}

void handle_cudaMemcpy(struct cudaMemcpyArg *arg)
{
    struct cudaMemcpyResponse *res;
    size_t res_size = 0;

    if (arg->kind == hipMemcpyHostToDevice) {

        // Only care about dst (pointer to device memory) from the client call
        hipError_t err = hipMemcpy(arg->dst, arg->data, arg->count, arg->kind);

        res_size = sizeof(hipError_t);
        res = (struct cudaMemcpyResponse *) malloc(res_size);
        res->err = err;
    } else if (arg->kind == hipMemcpyDeviceToHost){
        res_size = sizeof(hipError_t) + arg->count;
        res = (struct cudaMemcpyResponse *) malloc(res_size);

        // Only care about src (pointer to device memory) from the client call
        hipError_t err = hipMemcpy(res->data, arg->src, arg->count, arg->kind);

        res->err = err;
    } else {
        throw std::runtime_error("Unknown memcpy kind!");
    }

    while(!send_ipc->send((void *) res, res_size)) {
        send_ipc->wait_for_recv(1);
    }
}

void handle_cudaLaunchKernel(cudaLaunchKernelArg *arg)
{
    static hipError_t (*lcudaLaunchKernel) (const void *, dim3 , dim3 , void **, size_t , hipStream_t );
    if (!lcudaLaunchKernel) {
        lcudaLaunchKernel = (hipError_t (*) (const void *, dim3 , dim3 , void **, size_t , hipStream_t )) dlsym(RTLD_NEXT, "hipLaunchKernel");
    }
    assert(lcudaLaunchKernel);

    void *kernel_server_addr = _kernel_client_addr_mapping[(void *) arg->host_func];
    auto &arg_sizes = _kernel_addr_to_args[kernel_server_addr];
    auto argc = arg_sizes.size();

    void *__args_arr[argc];
    int __args_idx = 0;
    int offset = 0;

    for (size_t i = 0; i < argc; i++) {
        __args_arr[__args_idx] = (void *) (arg->params + offset);
        ++__args_idx;
        offset += arg_sizes[i];
    }

    auto err = lcudaLaunchKernel((const void *) kernel_server_addr, arg->gridDim, arg->blockDim, &__args_arr[0], arg->sharedMem, hipStreamDefault);

    while (!send_ipc->send((void *) &err, sizeof(hipError_t))) {
        send_ipc->wait_for_recv(1);
    }
}

void handle_fatCubin(fatBinArg *arg)
{
    magic = arg->magic;
    version = arg->version;

    struct fatBinaryHeader *fbh = (struct fatBinaryHeader *) arg->data;
    fatBinSize = fbh->headerSize + fbh->fatSize;

    fatbin_data = (unsigned long long *) malloc(fatBinSize);
    memcpy(fatbin_data, arg->data, fatBinSize);
}

void handle_register_kernel(struct registerKernelArg *arg)
{
    std::string kernel_name {arg->data, arg->kernel_func_len};
    register_queue.push_back( std::make_pair(arg->host_func, kernel_name));
}

void handle_fatCubin_end()
{
    static void (*l__cudaRegisterFatBinaryEnd) (void **);
	if (!l__cudaRegisterFatBinaryEnd) {
		l__cudaRegisterFatBinaryEnd = (void (*) (void **)) dlsym(RTLD_NEXT, "__cudaRegisterFatBinaryEnd");
	}
	assert(l__cudaRegisterFatBinaryEnd);
    static void** (*l__cudaRegisterFatBinary) (void *);
    if (!l__cudaRegisterFatBinary) {
        l__cudaRegisterFatBinary = (void** (*) (void *)) dlsym(RTLD_NEXT, "__cudaRegisterFatBinary");
    }
    assert(l__cudaRegisterFatBinary);
    static void (*l__cudaRegisterFunction) (void **, const char *, char *, const char *, int , uint3 *, uint3 *, dim3 *, dim3 *, int *);
    if (!l__cudaRegisterFunction) {
        l__cudaRegisterFunction = (void (*) (void **, const char *, char *, const char *, int , uint3 *, uint3 *, dim3 *, dim3 *, int *)) dlsym(RTLD_NEXT, "__cudaRegisterFunction");
    }
    assert(l__cudaRegisterFunction);

    const my__fatBinC_Wrapper_t __fatDeviceText __attribute__ ((aligned (8))) = { magic, version, fatbin_data, 0 };
    
    void **handle = l__cudaRegisterFatBinary((void *)&__fatDeviceText);

    void *kernel_server_addr;

    for (auto &kernel_pair : register_queue) {
        auto &client_addr = kernel_pair.first;
        auto &kernel_name = kernel_pair.second;

        // allocate an address for the purpose
        kernel_server_addr = malloc(8);

        // Bookkeeping the mapping between clinet kernel addr and server kernel addr
        _kernel_name_to_addr[kernel_name] = kernel_server_addr;
        _kernel_client_addr_mapping[client_addr] = kernel_server_addr;
        l__cudaRegisterFunction(handle, (const char*) kernel_server_addr, (char *)kernel_name.c_str(), kernel_name.c_str(), -1, (uint3*)0, (uint3*)0, (dim3*)0, (dim3*)0, (int*)0);
    }

    l__cudaRegisterFatBinaryEnd(handle);

    std::ofstream cubin_file("/tmp/tmp.cubin", std::ios::binary); // Open the file in binary mode
    cubin_file.write(reinterpret_cast<const char*>(fatbin_data), fatBinSize);
    cubin_file.close();

    const char* command = "cuobjdump /tmp/tmp.cubin -elf > /tmp/tmp_cubin.elf";
    system(command);

    std::string filename = "/tmp/tmp_cubin.elf";
    std::ifstream elf_file(filename);

    // key: func_name, val: [ <ordinal, size> ]
    using ordinal_size_pair = std::pair<uint32_t, uint32_t>;

    std::string line;
    while (std::getline(elf_file, line)) {
        if (startsWith(line, ".nv.info.")) {
            std::string kernel_name = line.substr(9);
            std::vector<ordinal_size_pair> params_info;

            while (std::getline(elf_file, line)) {
                if (containsSubstring(line, "EIATTR_KPARAM_INFO")) {
                    
                } else if (containsSubstring(line, "Ordinal :")) {
                    auto split_by_ordinal = splitOnce(line, "Ordinal :");
                    auto split_by_offset = splitOnce(split_by_ordinal.second, "Offset  :");
                    auto split_by_size = splitOnce(split_by_offset.second, "Size    :");

                    auto ordinal_str = strip(split_by_offset.first);
                    auto size_str = strip(split_by_size.second);

                    uint32_t arg_ordinal = std::stoi(ordinal_str, nullptr, 16);
                    uint32_t arg_size = std::stoi(size_str, nullptr, 16);

                    params_info.push_back(std::make_pair(arg_ordinal, arg_size));

                } else if (line.empty()) {
                    break;
                }
            }

            // Sort by ordinal
            std::sort(
                params_info.begin(),
                params_info.end(),
                [](ordinal_size_pair a, ordinal_size_pair b) {
                    return a.first < b.first;
                }
            );

            // Store the size
            for (auto &pair : params_info) {
                _kernel_addr_to_args[_kernel_name_to_addr[kernel_name]].push_back(pair.second);
            }
        }
    }    

    elf_file.close();

    // For some reason, must call one cuda api call here. Otherwise it won't run.
    int *arr;
    hipMalloc((void**)&arr, sizeof(int));
    hipFree(arr);
}

void do_recv(int interval) {
    send_ipc = new ipc::channel("server-to-client", ipc::sender);
    recv_ipc = new ipc::channel("client-to-server", ipc::receiver);

    while (!is_quit__.load(std::memory_order_acquire)) {
        ipc::buff_t buf;
        while (buf.empty()) {
            buf = recv_ipc->recv(interval);
            if (is_quit__.load(std::memory_order_acquire)) return;
        }

        char const *dat = buf.get<char const *>();
        
        int func_name_length = *((int *)dat);
        std::string func_name(dat + 4, func_name_length);
        std::cout << func_name << std::endl;

        void *arg_sizes = (void *) (dat + 4 + func_name_length);

        if (func_name == "hipMalloc") {
            handle_cudaMalloc((cudaMallocArg *) arg_sizes);
        } else if (func_name == "hipMemcpy") {
            handle_cudaMemcpy((cudaMemcpyArg *) arg_sizes);
        } else if (func_name == "hipLaunchKernel") {
            handle_cudaLaunchKernel((cudaLaunchKernelArg *) arg_sizes);
        } else if (func_name == "__cudaRegisterFunction") {
            handle_register_kernel((registerKernelArg *) arg_sizes);
        } else if (func_name == "__cudaRegisterFatBinary") {
            handle_fatCubin((fatBinArg *) arg_sizes);
        } else if (func_name == "__cudaRegisterFatBinaryEnd") {
            handle_fatCubin_end();
        }
    }
}

int main(int argc, char ** argv) {

    auto _exit = [](int) {
        is_quit__.store(true, std::memory_order_release);
        if (send_ipc != nullptr) send_ipc->disconnect();
        if (recv_ipc != nullptr) recv_ipc->disconnect();
        exit(0);
    };

    signal(SIGINT  , _exit);
    signal(SIGABRT , _exit);
    signal(SIGSEGV , _exit);
    signal(SIGTERM , _exit);
    signal(SIGHUP  , _exit);

    do_recv(1000);

    return 0;
}