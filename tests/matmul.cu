
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <sys/time.h>

__global__ void matrixMultiply(float *A, float *B, float *C, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

__host__ void runmatrixMultiply(float *h_A, float *h_B, float *h_C, int width)
{
    int size = width * width * sizeof(float);
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, width);

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void runmatrixMultiplyCpu(float *h_A, float *h_B, float *h_C, int width)
{
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < width; ++k) {
                sum += h_A[i * width + k] * h_B[k * width + j];
            }
            h_C[i * width + j] = sum;
        }
    }
}

int main()
{
    int width = 64;
    float h_A[64][64];
    float h_B[64][64];
    float h_C_cpu[64][64];
    float h_C_gpu[64][64];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            h_A[i][j] = static_cast<float>(std::rand()) / RAND_MAX;
            h_B[i][j] = static_cast<float>(std::rand()) / RAND_MAX;
        }
    }

    runmatrixMultiply((float *)h_A, (float *)h_B, (float *)h_C_gpu, width);
    runmatrixMultiplyCpu((float *)h_A, (float *)h_B, (float *)h_C_cpu, width);

    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            assert((h_C_gpu[i][j] - h_C_cpu[i][j]) < 0.0001);
        }
    }

    return 0;
}
