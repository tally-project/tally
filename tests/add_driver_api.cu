#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void addKernel(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int numElements = 1024;
    const int blockSize = 256;
    const int gridSize = (numElements + blockSize - 1) / blockSize;

    // Initialize CUDA Driver API
    hipInit(0);

    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);

    hipCtx_t cuContext;
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Allocate device memory
    hipDeviceptr_t d_a, d_b, d_c;
    hipMallocAsync(&d_a, numElements * sizeof(float), NULL);
    hipMallocAsync(&d_b, numElements * sizeof(float), NULL);
    hipMallocAsync(&d_c, numElements * sizeof(float), NULL);

    // Initialize host arrays
    float *h_a = new float[numElements];
    float *h_b = new float[numElements];
    float *h_c = new float[numElements];
    for (int i = 0; i < numElements; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Copy host data to device
    cuMemcpy(d_a, (hipDeviceptr_t)h_a, numElements * sizeof(float));
    cuMemcpy(d_b, (hipDeviceptr_t)h_b, numElements * sizeof(float));

    // Launch kernel
    addKernel<<<gridSize, blockSize>>>((float *)d_a, (float *)d_b, (float *)d_c, numElements);

    // Copy result back to host
    cuMemcpy((hipDeviceptr_t)h_c, d_c, numElements * sizeof(float));

    // Print some results
    for (int i = 0; i < 10; ++i) {
        printf("h_c[%d] = %f\n", i, h_c[i]);
    }

    // Clean up
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipCtxDestroy(cuContext);

    return 0;
}
