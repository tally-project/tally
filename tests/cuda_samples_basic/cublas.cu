#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>

#include <hipblas.h>
#include "hip/hip_runtime.h"

int main()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int m = 1024;
    int n = 1024;
    int k = 1024;

    float *h_A, *h_B, *h_C;
    h_A = (float *) malloc(sizeof(float) * m * k);
    h_B = (float *) malloc(sizeof(float) * k * n);
    h_C = (float *) malloc(sizeof(float) * m * n);

    // Allocate memory on the device
    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(float) * m * k); // m x k matrix
    hipMalloc((void**)&d_B, sizeof(float) * k * n); // k x n matrix
    hipMalloc((void**)&d_C, sizeof(float) * m * n); // m x n matrix

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * k * n, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasOperation_t transa = HIPBLAS_OP_N; // No transpose for A
    hipblasOperation_t transb = HIPBLAS_OP_N; // No transpose for B
    int lda = m; // Leading dimension of A (A is a m x k matrix)
    int ldb = k; // Leading dimension of B (B is a k x n matrix)
    int ldc = m; // Leading dimension of C (C is a m x n matrix)

    hipblasSgemm(handle, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);

    hipMemcpy(h_C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}