#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>
#include <chrono>

#include <hipblas.h>
#include <hipblaslt.h>

#include <tally/cutlass/cutlass.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main()
{
    srand (1);

    int m = 768;
    int n = 51865;
    int k = 1376;

    int lda = 768;
    int ldb = 51865;
    int ldc = 768;

    bool use_fp16 = true;

    std::cout << "M: " << m << " N: " << n << " K: " << k << std::endl;  
    std::cout << "use_fp16: " << use_fp16 << std::endl;  

    void *h_A, *h_B;
    void *h_cublas, *h_cutlass;

    size_t A_size_bytes;
    size_t B_size_bytes;
    size_t C_size_bytes;

    if (use_fp16) {
        A_size_bytes = sizeof(half) * m * k;
        B_size_bytes = sizeof(half) * k * n;
        C_size_bytes = sizeof(half) * m * n;
    } else {
        A_size_bytes = sizeof(float) * m * k;
        B_size_bytes = sizeof(float) * k * n;
        C_size_bytes = sizeof(float) * m * n;
    }

    // A will be k * m
    h_A = malloc(A_size_bytes);
    // B will be k * n
    h_B = malloc(B_size_bytes);

    h_cublas = malloc(C_size_bytes);
    h_cutlass = malloc(C_size_bytes);

    // Set values in h_A
    for (int j = 0; j < m; j++) {
        for (int i = 0; i < k; i++) {
            float val = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

            if (use_fp16) {
                ((half *)h_A)[IDX2C(i, j, k)] = __float2half(val);
            } else {
                ((float *)h_A)[IDX2C(i, j, k)] = val;
            }
        }
    }

    // Set values in h_B
    for (int j = 0; j < n; j++) {
        for (int i = 0; i < k; i++) {
            float val = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

            if (use_fp16) {
                ((half *)h_B)[IDX2C(i, j, k)] = __float2half(val);
            } else {
                ((float *)h_B)[IDX2C(i, j, k)] = val;
            }
        }
    }

    // Allocate memory on the device
    void* d_A, *d_B;
    float *d_cublas, *d_cutlass;
    hipMalloc(&d_A, A_size_bytes);
    hipMalloc(&d_B, B_size_bytes);
    hipMalloc(&d_cublas, C_size_bytes);
    hipMalloc(&d_cutlass, C_size_bytes);
    hipMemset(d_cublas, 0, C_size_bytes);
    hipMemset(d_cutlass, 0, C_size_bytes);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, A_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_size_bytes, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.f;

    hipblasOperation_t transa = HIPBLAS_OP_N; // No transpose for A
    hipblasOperation_t transb = HIPBLAS_OP_N; // No transpose for B

    cutlassOperation_t transa_cutlass = CUTLASS_OP_N;
    cutlassOperation_t transb_cutlass = CUTLASS_OP_N;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasLtHandle_t lightHandle;
    hipblasLtCreate(&lightHandle);

    hipblasLtMatmulDesc_t matmul_desc;
    hipblasLtMatrixLayout_t A_layout;
    hipblasLtMatrixLayout_t B_layout;
    hipblasLtMatrixLayout_t C_layout;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult;

    hipblasLtMatmulDescCreate(&matmul_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

    hipblasLtMatrixLayoutCreate(&A_layout, HIP_R_16F, m, k, m);
    hipblasLtMatrixLayoutCreate(&B_layout, HIP_R_16F, k, n, k);
    hipblasLtMatrixLayoutCreate(&C_layout, HIP_R_16F, m, n, m);

    hipblasLtMatmulPreference_t preference = NULL;
    hipblasLtMatmulPreferenceCreate(&preference);

    hipblasLtMatmulAlgoGetHeuristic(lightHandle, matmul_desc, A_layout, B_layout, C_layout, C_layout, preference, 1, &heuristicResult, &returnedResults);

    // warmup
    if (use_fp16) {
        hipblasLtMatmul(lightHandle, matmul_desc, &alpha, d_A, A_layout, d_B, B_layout, &beta, d_cublas, C_layout, d_cublas, C_layout, &heuristicResult.algo, NULL, 0, 0);
        cutlassGemm_f16(transa_cutlass, transb_cutlass, m, n, k, alpha, (half *)d_A, lda /*lda*/, (half *)d_B, ldb /*ldb*/, beta, (half *)d_cutlass, ldc /*ldc*/, (half *)d_cutlass, ldc /*ldd*/, NULL, NULL);
    } else {
        hipblasSgemm(handle, transa, transb, m, n, k, &alpha, (float *)d_A, lda /*lda*/, (float *)d_B, ldb /*ldb*/, &beta, d_cublas, ldc /*ldc*/);
        cutlassGemm_f32(transa_cutlass, transb_cutlass, m, n, k, alpha, (float *)d_A, lda /*lda*/, (float *)d_B, ldb /*ldb*/, beta, (float *)d_cutlass, ldc /*ldc*/, (float *)d_cutlass, ldc /*ldd*/, NULL, NULL, NULL);
    }

    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();

    if (use_fp16) {
        hipblasLtMatmul(lightHandle, matmul_desc, &alpha, d_A, A_layout, d_B, B_layout, &beta, d_cublas, C_layout, d_cublas, C_layout, &heuristicResult.algo, NULL, 0, 0);
    } else {
        hipblasSgemm(handle, transa, transb, m, n, k, &alpha, (float *)d_A, lda /*lda*/, (float *)d_B, ldb /*ldb*/, &beta, d_cublas, ldc /*ldc*/);
    }

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;
    auto cublas_ms = duration.count();

    hipDeviceSynchronize();
    start = std::chrono::high_resolution_clock::now();

    // Run cutlass impl
    if (use_fp16) {
        cutlassGemm_f16(transa_cutlass, transb_cutlass, m, n, k, alpha, (half *)d_A, lda /*lda*/, (half *)d_B, ldb /*ldb*/, beta, (half *)d_cutlass, ldc /*ldc*/, (half *)d_cutlass, ldc /*ldd*/, NULL, NULL);
    } else {
       cutlassGemm_f32(transa_cutlass, transb_cutlass, m, n, k, alpha, (float *)d_A, lda /*lda*/, (float *)d_B, ldb /*ldb*/, beta, (float *)d_cutlass, ldc /*ldc*/, (float *)d_cutlass, ldc /*ldd*/, NULL, NULL, NULL);
    }

    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    auto cutlass_ms = duration.count();

    std::cout << "cutlass: " << std::to_string(cutlass_ms) << "ms" << std::endl;
    std::cout << "cublas: " << std::to_string(cublas_ms) << "ms" << std::endl;

    hipMemcpy(h_cutlass, d_cutlass, C_size_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_cublas, d_cublas, C_size_bytes, hipMemcpyDeviceToHost);

    for (int j = 0; j < n; j++) {
        for (int i = 0; i < m; i++) {

            float cublas_val;
            float cutlass_val;

            if (use_fp16) {
                cublas_val = __half2float(((half *)h_cublas)[i * n + j]);
                cutlass_val = __half2float(((half *)h_cutlass)[i * n + j]);
            } else {
                cublas_val = ((float *)h_cublas)[i * n + j];
                cutlass_val = ((float *)h_cutlass)[i * n + j];
            }

            if (abs(cublas_val - cutlass_val) > 1) {
                std::cout << "Results do not match." << std::endl;
                std::cout << "idx: " << (i * n + j) << std::endl;
                std::cout << "cublas_val: " << cublas_val << std::endl;
                std::cout << "cutlass_val: " << cutlass_val << std::endl;
                exit(1);
            }

        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_cublas);
    hipFree(d_cutlass);
    hipblasDestroy(handle);

    return 0;
}