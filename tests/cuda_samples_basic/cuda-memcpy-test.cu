#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void kernel(int *d_data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        d_data[tid] = tid;
    }
}

int main() {
    int h_data[N];  // Host array
    int *d_data;    // Device array
    hipCtx_t cuContext;
    hipDeviceptr_t d_data_ptr;
    hipDevice_t cuDevice;

    // Initialize CUDA
    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Allocate device memory
    hipMalloc(&d_data_ptr, N * sizeof(int));
    d_data = (int*)d_data_ptr;

    // Launch kernel to fill device array
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    kernel<<<grid, block>>>(d_data, N);
    hipCtxSynchronize(); // Wait for the kernel to finish

    // Copy data from device to host asynchronously
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);
    hipMemcpyDtoHAsync(h_data, d_data_ptr, N * sizeof(int), stream);

    // Synchronize the stream to make sure the copy is complete
    hipStreamSynchronize(stream);

    // Verify the data
    for (int i = 0; i < N; i++) {
        if (h_data[i] != i) {
            fprintf(stderr, "Verification failed at element %d!\n", i);
            return 1;
        }
    }

    // Clean up
    // hipStreamDestroy(stream);
    // hipFree(d_data_ptr);
    // hipCtxDestroy(cuContext);

    printf("Test completed successfully!\n");
    return 0;
}

