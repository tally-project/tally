#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>
#include <chrono>
#include <thread>
#include <unistd.h>

#include "hip/hip_runtime.h"

__global__ void elementwiseAddition(float* a, float* b, float* c, int size) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < size && tid % 2 == 0) {
        c[tid] = a[tid] + b[tid];
    }
}

void checkCudaErrors(hipError_t err) {

    if (err) {
        char *str;

        hipDrvGetErrorString(err, (const char**)&str);

        std::cout << str << std::endl;
    }

  assert(err == hipSuccess);
}

__host__ void runElementwiseAddition(float* arr_a, float* arr_b, float* arr_c, int size, bool ptb)
{
    // Allocate memory on the device (GPU)
    float* deviceA, * deviceB, * deviceC;
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));

    // Copy input arrays from host to device
    hipMemcpy(deviceA, arr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, arr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 block_dim(256);
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

    elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);

    hipMemcpy(arr_c, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void runElementwiseAdditionCpu(float* arr_a, float* arr_b, float* arr_c, int size)
{
    for (size_t i = 0; i < size; i++) {

        if (i % 2 == 0) {
            arr_c[i] = arr_a[i] + arr_b[i];
        }
    }
}

int main()
{
    int size = 16777216;
    bool ptb = false;
    
    // Allocate memory on the host (CPU)
    float* arr_a = new float[size];
    float* arr_b = new float[size];
    float* res_gpu = new float[size];
    float* res_cpu = new float[size];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }
    
    runElementwiseAddition(arr_a, arr_b, res_gpu, size, ptb);
    runElementwiseAdditionCpu(arr_a, arr_b, res_cpu, size);

    for (int i = 0; i < size; i++) {
        if (abs(res_gpu[i] - res_cpu[i]) > 0.0001) {
            std::cerr << "Index i: " << i << " result mismatch: res_gpu[i]: " << res_gpu[i] << " " << "res_cpu[i]: " << res_cpu[i] << std::endl;
            exit(1);
        }
    }
    
    // Cleanup
    delete[] arr_a;
    delete[] arr_b;
    delete[] res_cpu;
    delete[] res_gpu;

    return 0;
}