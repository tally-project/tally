#include <nccl.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int main() {
    // Initialize NCCL
    ncclComm_t comm;
    hipStream_t stream;
    hipStreamCreate(&stream);
    ncclUniqueId id;
    ncclGetUniqueId(&id);
    ncclCommInitRank(&comm, 1, id, 0);

    // Allocate host and device buffers
    int N = 1024;
    float *sendbuff, *recvbuff;
    hipMalloc(&sendbuff, N * sizeof(float));
    hipMalloc(&recvbuff, N * sizeof(float));

    // Initialize send buffer with some values
    std::vector<float> host_data(N, 1.0f); // Fill with 1.0f for example
    hipMemcpy(sendbuff, host_data.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // Perform all-reduce operation
    ncclAllReduce(sendbuff, recvbuff, N, ncclFloat, ncclSum, comm, stream);

    // Wait for the operation to complete
    hipStreamSynchronize(stream);

    // Copy result back to host and print it
    std::vector<float> result(N);
    hipMemcpy(result.data(), recvbuff, N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        std::cout << result[i] << " ";
    }
    std::cout << std::endl;

    // Finalize
    hipFree(sendbuff);
    hipFree(recvbuff);
    ncclCommDestroy(comm);
    hipStreamDestroy(stream);

    return 0;
}
