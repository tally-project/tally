#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>
#include <chrono>
#include <thread>
#include <unistd.h>

#include "hip/hip_runtime.h"

__global__ void matrixMultiply(float *A, float *B, float *C, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

__global__ void elementwiseAddition(float* a, float* b, float* c, int size) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

__host__ void run_elem_matmul(float* arr_a, float* arr_b, float* arr_c, int size, bool ptb)
{
    int width = 4096;
    int matmul_size = width * width * sizeof(float);
    float *d_A, *d_B, *d_C;
    float* h_A = new float[width * width];
    float* h_B = new float[width * width];
    float* res_gpu = new float[width * width];
    float* res_cpu = new float[width * width];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < width * width; ++i) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, matmul_size);
    hipMalloc((void **)&d_B, matmul_size);
    hipMalloc((void **)&d_C, matmul_size);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, matmul_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matmul_size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);
    std::cout << "gridSize: " << gridSize.x << " " << gridSize.y << " " << gridSize.z << std::endl;


    // Allocate memory on the device (GPU)
    float* deviceA, * deviceB, * deviceC;
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));

    // Copy input arrays from host to device
    hipMemcpy(deviceA, arr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, arr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 block_dim(256);
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

    // same as before
    dim3 PTB_block_dim(256);

    // Depend on number of PTBs/SM
    dim3 PTB_grid_dim(82 * 4);

    bool *retreat;
    hipMalloc((void**)&retreat, sizeof(bool));

    uint32_t *global_idx;
    hipMalloc((void**)&global_idx, sizeof(uint32_t));

    hipStream_t kernel_stream;
    hipStreamCreate(&kernel_stream);

    hipStream_t new_stream;
    hipStreamCreate(&new_stream);

    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    for (int i = 0; i < 150; i++) {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
        matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, width);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel execution time: %.2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(arr_c, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void run_elem_matmulCpu(float* arr_a, float* arr_b, float* arr_c, int size)
{
    for (size_t i = 0; i < size; i++) {
        arr_c[i] = arr_a[i] + arr_b[i];
    }
}

int main()
{
    int size = 134217728;
    bool ptb = false;
    
    // Allocate memory on the host (CPU)
    float* arr_a = new float[size];
    float* arr_b = new float[size];
    float* res_gpu = new float[size];
    float* res_cpu = new float[size];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }
    
    run_elem_matmul(arr_a, arr_b, res_gpu, size, ptb);
    
    // Cleanup
    delete[] arr_a;
    delete[] arr_b;
    delete[] res_cpu;
    delete[] res_gpu;

    return 0;
}