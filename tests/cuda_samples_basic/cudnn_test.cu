#include <iostream>
#include <hip/hip_runtime.h>
#include <cudnn.h>

/**
 * Minimal example to apply sigmoid activation on a tensor 
 * using cuDNN.
 **/
int main(int argc, char** argv)
{    
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Found " << numGPUs << " GPUs." << std::endl;
    hipSetDevice(0); // use GPU0
    int device; 
    struct hipDeviceProp_t devProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&devProp, device);
    std::cout << "Compute capability:" << devProp.major << "." << devProp.minor << std::endl;

    cudnnHandle_t handle_;
    cudnnCreate(&handle_);
    std::cout << "Created cuDNN handle" << std::endl;

    // create the tensor descriptor
    cudnnDataType_t dtype = CUDNN_DATA_FLOAT;
    cudnnTensorFormat_t format = CUDNN_TENSOR_NCHW;
    int n = 1, c = 1, h = 1, w = 10;
    int NUM_ELEMENTS = n*c*h*w;
    cudnnTensorDescriptor_t x_desc;
    cudnnCreateTensorDescriptor(&x_desc);
    cudnnSetTensor4dDescriptor(x_desc, format, dtype, n, c, h, w);

    // create the tensor
    float *x = (float *)malloc(NUM_ELEMENTS * sizeof(float));

    float *dev_ptr;
    hipMalloc(&dev_ptr, NUM_ELEMENTS * sizeof(float));

    for(int i=0;i<NUM_ELEMENTS;i++) x[i] = i * 1.00f;
    std::cout << "Original array: "; 
    for(int i=0;i<NUM_ELEMENTS;i++) std::cout << x[i] << " ";

    hipMemcpy(dev_ptr, x, NUM_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);

    // create activation function descriptor
    float alpha[1] = {1};
    float beta[1] = {0.0};
    cudnnActivationDescriptor_t sigmoid_activation;
    cudnnActivationMode_t mode = CUDNN_ACTIVATION_SIGMOID;
    cudnnNanPropagation_t prop = CUDNN_NOT_PROPAGATE_NAN;
    cudnnCreateActivationDescriptor(&sigmoid_activation);
    cudnnSetActivationDescriptor(sigmoid_activation, mode, prop, 0.0f);

    cudnnActivationForward(
        handle_,
        sigmoid_activation,
        alpha,
        x_desc,
        dev_ptr,
        beta,
        x_desc,
        dev_ptr
    );

    cudnnDestroy(handle_);
    std::cout << std::endl << "Destroyed cuDNN handle." << std::endl;
    std::cout << "New array: ";

    hipMemcpy(x, dev_ptr, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0;i<NUM_ELEMENTS;i++) std::cout << x[i] << " ";
    std::cout << std::endl;
    hipFree(x);

    return 0;
}