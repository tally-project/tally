#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>

#include "hip/hip_runtime.h"

__global__ void elementwiseAddition(float* a, float* b, float* c, int size) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void elementwiseAdditionPTB(float* a, float* b, float* c, int size, dim3 original_gridSize) {

    uint32_t num_thread_blocks = original_gridSize.x * original_gridSize.y * original_gridSize.z;
    uint32_t xy_tbs = original_gridSize.x * original_gridSize.y;
    dim3 newBlockIdx(0, 0, 0);

    for (int tb_idx = blockIdx.x; tb_idx < num_thread_blocks; tb_idx += gridDim.x) {

        newBlockIdx.z = tb_idx / xy_tbs;
        newBlockIdx.y = (tb_idx - newBlockIdx.z * xy_tbs) / original_gridSize.x;
        newBlockIdx.x = (tb_idx - newBlockIdx.z * xy_tbs) - newBlockIdx.y * original_gridSize.x;

        int tid = threadIdx.x + newBlockIdx.x * blockDim.x;
    
        if (tid < size) {
            c[tid] = a[tid] + b[tid];
        }
    }
}

__host__ void runElementwiseAddition(float* arr_a, float* arr_b, float* arr_c, int size, bool ptb)
{
    // Allocate memory on the device (GPU)
    float* deviceA, * deviceB, * deviceC;
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));
    
    // Copy input arrays from host to device
    hipMemcpy(deviceA, arr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, arr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 block_dim(256);
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

    // same as before
    dim3 PTB_block_dim(256);

    // Depend on number of PTBs/SM
    dim3 PTB_grid_dim(82 * 4);

    // Warm up
    if (ptb) {
        elementwiseAdditionPTB<<<PTB_grid_dim, PTB_block_dim>>>(deviceA, deviceB, deviceC, size, grid_dim);
    } else {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
    }

    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Launch the kernel
    if (ptb) {
        elementwiseAdditionPTB<<<PTB_grid_dim, PTB_block_dim>>>(deviceA, deviceB, deviceC, size, grid_dim);
    } else {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel execution time: %.2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(arr_c, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void runElementwiseAdditionCpu(float* arr_a, float* arr_b, float* arr_c, int size)
{
    for (size_t i = 0; i < size; i++) {
        arr_c[i] = arr_a[i] + arr_b[i];
    }
}

int main()
{
    int size = 16777216;
    bool ptb = false;
    
    // Allocate memory on the host (CPU)
    float* arr_a = new float[size];
    float* arr_b = new float[size];
    float* res_gpu = new float[size];
    float* res_cpu = new float[size];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }
    
    runElementwiseAddition(arr_a, arr_b, res_gpu, size, ptb);
    runElementwiseAdditionCpu(arr_a, arr_b, res_cpu, size);

    for (int i = 0; i < size; i++) {
        if (abs(res_gpu[i] - res_cpu[i]) > 0.0001) {
            std::cerr << "result mismatch: res_gpu[i]: " << res_gpu[i] << " " << "res_cpu[i]: " << res_gpu[i] << std::endl;
            exit(1);
        }
    }
    
    // Cleanup
    delete[] arr_a;
    delete[] arr_b;
    delete[] res_cpu;
    delete[] res_gpu;

    return 0;
}