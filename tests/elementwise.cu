#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>

#include "hip/hip_runtime.h"

__global__ void elementwiseAddition(float* a, float* b, float* c, int size) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void elementwiseAdditionPTB(float* a, float* b, float* c, int size, dim3 original_gridSize) {

    uint32_t num_thread_blocks = original_gridSize.x * original_gridSize.y * original_gridSize.z;
    uint32_t xy_tbs = original_gridSize.x * original_gridSize.y;
    dim3 newBlockIdx(0, 0, 0);

    for (int tb_idx = blockIdx.x; tb_idx < num_thread_blocks; tb_idx += gridDim.x) {

        newBlockIdx.z = tb_idx / xy_tbs;
        newBlockIdx.y = (tb_idx - newBlockIdx.z * xy_tbs) / original_gridSize.x;
        newBlockIdx.x = (tb_idx - newBlockIdx.z * xy_tbs) - newBlockIdx.y * original_gridSize.x;

        int tid = threadIdx.x + newBlockIdx.x * blockDim.x;
    
        if (tid < size) {
            c[tid] = a[tid] + b[tid];
        }
    }
}

__device__ volatile bool retreat = 0;
__device__ unsigned int global_idx = 0;

__global__ void elementwiseAdditionPTB_dynamic(float* a, float* b, float* c, int size, dim3 original_gridSize) {

    const bool leader = (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);
    const uint32_t num_thread_blocks = original_gridSize.x * original_gridSize.y * original_gridSize.z;
    const uint32_t xy_tbs = original_gridSize.x * original_gridSize.y;

    __shared__ volatile unsigned int curr_idx;

    while (true) {

        if (leader) {
            if (retreat) {
                curr_idx = num_thread_blocks + 1;
            } else {
                curr_idx = atomicAdd(&global_idx, 1);
            }
        }

        __syncthreads();

        if (curr_idx > num_thread_blocks) {
            break;
        }

        dim3 newBlockIdx(0, 0, 0);

        newBlockIdx.z = curr_idx / xy_tbs;
        newBlockIdx.y = (curr_idx - newBlockIdx.z * xy_tbs) / original_gridSize.x;
        newBlockIdx.x = (curr_idx - newBlockIdx.z * xy_tbs) - newBlockIdx.y * original_gridSize.x;

        int tid = threadIdx.x + newBlockIdx.x * blockDim.x;
    
        if (tid < size) {
            c[tid] = a[tid] + b[tid];
        }
    }
}

__host__ void runElementwiseAddition(float* arr_a, float* arr_b, float* arr_c, int size, bool ptb)
{
    // Allocate memory on the device (GPU)
    float* deviceA, * deviceB, * deviceC;
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));
    
    // Copy input arrays from host to device
    hipMemcpy(deviceA, arr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, arr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 block_dim(256);
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

    // same as before
    dim3 PTB_block_dim(256);

    // Depend on number of PTBs/SM
    dim3 PTB_grid_dim(82 * 4);

    void *global_idx_devptr;
    hipGetSymbolAddress(&global_idx_devptr, HIP_SYMBOL(global_idx));

    void *retreat_devptr;
    hipGetSymbolAddress(&retreat_devptr, HIP_SYMBOL(retreat));

    hipStream_t kernel_stream;
    hipStreamCreate(&kernel_stream);

    hipStream_t new_stream;
    hipStreamCreate(&new_stream);

    // // Warm up
    if (ptb) {
        hipMemset(global_idx_devptr, 0, sizeof(int));
        elementwiseAdditionPTB_dynamic<<<PTB_grid_dim, PTB_block_dim>>>(deviceA, deviceB, deviceC, size, grid_dim);
    } else {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
    }

    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Launch the kernel
    if (ptb) {
        // elementwiseAdditionPTB<<<PTB_grid_dim, PTB_block_dim>>>(deviceA, deviceB, deviceC, size, grid_dim);

        hipMemset(retreat_devptr, 0, sizeof(bool));
        hipMemset(global_idx_devptr, 0, sizeof(int));

        elementwiseAdditionPTB_dynamic<<<PTB_grid_dim, PTB_block_dim, 0, kernel_stream>>>(deviceA, deviceB, deviceC, size, grid_dim);

        hipMemsetAsync(retreat_devptr, 1, sizeof(bool), new_stream);

        int progress = 0;
        hipMemcpy(&progress, global_idx_devptr, sizeof(int), hipMemcpyDeviceToHost);

        // std::cout << "progress: " << progress << std::endl; 

        hipMemsetAsync(retreat_devptr, 0, sizeof(bool), new_stream);

        elementwiseAdditionPTB_dynamic<<<PTB_grid_dim, PTB_block_dim, 0, kernel_stream>>>(deviceA, deviceB, deviceC, size, grid_dim);

        // cudaMemcpy(&progress, global_idx_devptr, sizeof(int), cudaMemcpyDeviceToHost);

        // std::cout << "progress: " << progress << std::endl; 

    } else {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel execution time: %.2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(arr_c, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void runElementwiseAdditionCpu(float* arr_a, float* arr_b, float* arr_c, int size)
{
    for (size_t i = 0; i < size; i++) {
        arr_c[i] = arr_a[i] + arr_b[i];
    }
}

int main()
{
    int size = 16777216;
    bool ptb = true;
    
    // Allocate memory on the host (CPU)
    float* arr_a = new float[size];
    float* arr_b = new float[size];
    float* res_gpu = new float[size];
    float* res_cpu = new float[size];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }
    
    runElementwiseAddition(arr_a, arr_b, res_gpu, size, ptb);
    runElementwiseAdditionCpu(arr_a, arr_b, res_cpu, size);

    for (int i = 0; i < size; i++) {
        if (abs(res_gpu[i] - res_cpu[i]) > 0.0001) {
            std::cerr << "Index i: " << i << " result mismatch: res_gpu[i]: " << res_gpu[i] << " " << "res_cpu[i]: " << res_cpu[i] << std::endl;
            exit(1);
        }
    }
    
    // Cleanup
    delete[] arr_a;
    delete[] arr_b;
    delete[] res_cpu;
    delete[] res_gpu;

    return 0;
}