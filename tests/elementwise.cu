#include <stdio.h>
#include <cassert>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <sys/time.h>

#include "hip/hip_runtime.h"

__global__ void elementwiseAddition(float* a, float* b, float* c, int size) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void elementwiseAdditionPTB(float* a, float* b, float* c, int size, dim3 original_gridSize) {

    uint32_t part_size = original_gridSize.x / gridDim.x;
    uint32_t start_idx = blockIdx.x * part_size;
    uint32_t end_idx = start_idx + part_size;
    if (blockIdx.x == gridDim.x - 1) {
        end_idx = original_gridSize.x;
    }

    for (int curr_block_idx = start_idx; curr_block_idx < end_idx; curr_block_idx++) {

        if (curr_block_idx < original_gridSize.x) {

            int tid = threadIdx.x + curr_block_idx * blockDim.x;

            if (tid < size) {
                c[tid] = a[tid] + b[tid];
            }

        }
    }
}

__host__ void runElementwiseAddition(float* arr_a, float* arr_b, float* arr_c, int size, bool ptb)
{
    // Allocate memory on the device (GPU)
    float* deviceA, * deviceB, * deviceC;
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));
    
    // Copy input arrays from host to device
    hipMemcpy(deviceA, arr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, arr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 block_dim(256);
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

    // same as before
    dim3 PTB_block_dim(256);

    // Depend on number of PTBs/SM
    dim3 PTB_grid_dim(82 * 4);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Launch the kernel
    if (ptb) {
        elementwiseAdditionPTB<<<PTB_grid_dim, PTB_block_dim>>>(deviceA, deviceB, deviceC, size, grid_dim);
    } else {
        elementwiseAddition<<<grid_dim, block_dim>>>(deviceA, deviceB, deviceC, size);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel execution time: %.2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(arr_c, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void runElementwiseAdditionCpu(float* arr_a, float* arr_b, float* arr_c, int size)
{
    for (size_t i = 0; i < size; i++) {
        arr_c[i] = arr_a[i] + arr_b[i];
    }
}

int main()
{
    int size = 262144;
    bool ptb = false;
    
    // Allocate memory on the host (CPU)
    float* arr_a = new float[size];
    float* arr_b = new float[size];
    float* res_gpu = new float[size];
    float* res_cpu = new float[size];

    std::srand(std::time(nullptr));
    
    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        arr_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
        arr_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }
    
    runElementwiseAddition(arr_a, arr_b, res_gpu, size, ptb);
    runElementwiseAdditionCpu(arr_a, arr_b, res_cpu, size);

    for (int i = 0; i < size; i++) {
        assert(res_gpu[i] == res_cpu[i]);
    }
    
    // Cleanup
    delete[] arr_a;
    delete[] arr_b;
    delete[] res_cpu;
    delete[] res_gpu;

    return 0;
}