
#include "cutlass/gemm/device/gemm.h"

#include <tally/cutlass/cutlass.h>

extern "C" {

void tally_register_cutlass()
{
    std::cout << "tally register cutlass ..." << std::endl;
}

hipError_t CutlassSgemmNN(
    cutlassOperation_t transA,
    cutlassOperation_t transB,
    int M,
    int N,
    int K,
    float alpha,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float beta,
    float *C,
    int ldc,
    void *workSpace,
    hipStream_t stream
) {

    using RowMajor = cutlass::layout::RowMajor;
    using ColumnMajor = cutlass::layout::ColumnMajor;

    cutlass::Status status;

    if (transA == cutlassOperation_t::CUTLASS_OP_N && transB == cutlassOperation_t::CUTLASS_OP_N) {

        using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                        ColumnMajor,  // Layout of A matrix
                                                        float,        // Data-type of B matrix
                                                        ColumnMajor,  // Layout of B matrix
                                                        float,        // Data-type of C matrix
                                                        ColumnMajor>; // Layout of C matrix

        CutlassGemm gemm_operator;

        CutlassGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    {A, lda},    // Tensor-ref for source matrix A
                                    {B, ldb},    // Tensor-ref for source matrix B
                                    {C, ldc},    // Tensor-ref for source matrix C
                                    {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                    {alpha, beta}); // Scalars used in the Epilogue


        status = gemm_operator(args, stream);

    } else if (transA == cutlassOperation_t::CUTLASS_OP_T && transB == cutlassOperation_t::CUTLASS_OP_N){
        using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                        RowMajor,  // Layout of A matrix
                                                        float,        // Data-type of B matrix
                                                        ColumnMajor,  // Layout of B matrix
                                                        float,        // Data-type of C matrix
                                                        ColumnMajor>; // Layout of C matrix

        CutlassGemm gemm_operator;

        CutlassGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    {A, lda},    // Tensor-ref for source matrix A
                                    {B, ldb},    // Tensor-ref for source matrix B
                                    {C, ldc},    // Tensor-ref for source matrix C
                                    {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                    {alpha, beta}); // Scalars used in the Epilogue


        status = gemm_operator(args, stream);
    } else if (transA == cutlassOperation_t::CUTLASS_OP_N && transB == cutlassOperation_t::CUTLASS_OP_T){
        using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                        ColumnMajor,  // Layout of A matrix
                                                        float,        // Data-type of B matrix
                                                        RowMajor,  // Layout of B matrix
                                                        float,        // Data-type of C matrix
                                                        ColumnMajor>; // Layout of C matrix

        CutlassGemm gemm_operator;

        CutlassGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    {A, lda},    // Tensor-ref for source matrix A
                                    {B, ldb},    // Tensor-ref for source matrix B
                                    {C, ldc},    // Tensor-ref for source matrix C
                                    {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                    {alpha, beta}); // Scalars used in the Epilogue


        status = gemm_operator(args, stream);
    } else if (transA == cutlassOperation_t::CUTLASS_OP_T && transB == cutlassOperation_t::CUTLASS_OP_T){
        using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                        RowMajor,  // Layout of A matrix
                                                        float,        // Data-type of B matrix
                                                        RowMajor,  // Layout of B matrix
                                                        float,        // Data-type of C matrix
                                                        ColumnMajor>; // Layout of C matrix

        CutlassGemm gemm_operator;

        CutlassGemm::Arguments args({M, N, K},  // Gemm Problem dimensions
                                    {A, lda},    // Tensor-ref for source matrix A
                                    {B, ldb},    // Tensor-ref for source matrix B
                                    {C, ldc},    // Tensor-ref for source matrix C
                                    {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                                    {alpha, beta}); // Scalars used in the Epilogue


        status = gemm_operator(args, stream);
    } else {
        throw std::runtime_error("Not implemented.");
    }

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

}