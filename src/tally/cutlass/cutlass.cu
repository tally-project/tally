#include <map>

#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_batched.h"
#include "cutlass/gemm/device/gemm_splitk_parallel.h"
#include "cutlass/util/host_tensor.h"

#include <tally/cutlass/cutlass.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

// Get the workspace with given size
// Will keep one workspace for each stream
void *get_workspace(size_t size, hipStream_t stream) {
    static std::map<hipStream_t, std::pair<size_t, void *>> workspace_map;

    if (workspace_map.find(stream) == workspace_map.end()) {
        workspace_map[stream].first = 0;
        workspace_map[stream].second = NULL;
    }

    if (size > workspace_map[stream].first) {
        hipStreamSynchronize(stream);
        auto old_workspace = workspace_map[stream].second;

        if (old_workspace) {
            hipFree(old_workspace);
        }

        void *workspace;
        hipMalloc(&workspace, size);
        workspace_map[stream].first = size;
        workspace_map[stream].second = workspace;
    }

    return workspace_map[stream].second;
}

using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    cutlass::half_t,
    128 / cutlass::sizeof_bits<cutlass::half_t>::value,
    float,
    float>;

#define CUTLASS_CHECK(status)                                                                    \
  {                                                                                              \
    cutlass::Status error = status;                                                              \
    if (error != cutlass::Status::kSuccess) {                                                    \
      std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) << " at: " << __LINE__ \
                << std::endl;                                                                    \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

#define CUTLASS_GEMM_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)  \
    using Gemm = cutlass::gemm::device::Gemm<ELEMENT_TYPE,                                      \
                                             LAYOUT_A,                                          \
                                             ELEMENT_TYPE,                                      \
                                             LAYOUT_B,                                          \
                                             ELEMENT_TYPE,                                      \
                                             LAYOUT_C,                                          \
                                             ELEMENT_ACCUMULATOR,                               \
                                             cutlass::arch::OpClassSimt,                        \
                                             cutlass::arch::Sm80,                               \
                                             cutlass::gemm::GemmShape<128, 128, 8>,             \
                                             cutlass::gemm::GemmShape<32, 64, 8>,               \
                                             cutlass::gemm::GemmShape<1, 1, 1>>;                \
    Gemm gemm_op;                                                                               \
    Gemm::Arguments args({M, N, K},                                                             \
                        {(ELEMENT_TYPE *) A, lda},                                              \
                        {(ELEMENT_TYPE *) B, ldb},                                              \
                        {(ELEMENT_TYPE *) C, ldc},                                              \
                        {(ELEMENT_TYPE *) D, ldd},                                              \
                        {alpha, beta});                                                         \
    status = gemm_op.can_implement(args);                                                       \
    if (status == cutlass::Status::kSuccess) {                                                  \
        size_t workspace_size = Gemm::get_workspace_size(args);                                 \
        void *workspace = get_workspace(workspace_size, stream);                                \
        gemm_op.initialize(args, workspace);                                                    \
        status = gemm_op(stream);                                                               \
    }

#define CUTLASS_GEMM_FP16_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)     \
    using Gemm = cutlass::gemm::device::Gemm<ELEMENT_TYPE,                                              \
                                             LAYOUT_A,                                                  \
                                             ELEMENT_TYPE,                                              \
                                             LAYOUT_B,                                                  \
                                             ELEMENT_TYPE,                                              \
                                             LAYOUT_C,                                                  \
                                             ELEMENT_ACCUMULATOR,                                       \
                                             cutlass::arch::OpClassTensorOp,                            \
                                             cutlass::arch::Sm80,                                       \
                                             cutlass::gemm::GemmShape<128, 128, 16>,                    \
                                             cutlass::gemm::GemmShape<64, 64, 16>,                      \
                                             cutlass::gemm::GemmShape<16, 8, 8>, \
                                             EpilogueOp, \
                                             cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, \
                                             4 /* NumStages */>;                       \
    Gemm gemm_op;                                                                                       \
    Gemm::Arguments args({M, N, K},                                                                     \
                        {(ELEMENT_TYPE *) A, lda},                                                      \
                        {(ELEMENT_TYPE *) B, ldb},                                                      \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        {(ELEMENT_TYPE *) D, ldd},                                                      \
                        {alpha, beta});                                                                 \
    status = gemm_op.can_implement(args);                                                               \
    if (status == cutlass::Status::kSuccess) {                                                  \
        size_t workspace_size = Gemm::get_workspace_size(args);                                             \
        void *workspace = get_workspace(workspace_size, stream);                                            \
        gemm_op.initialize(args, workspace);                                                                \
        status = gemm_op(stream);                                                                           \
    }

#define CUTLASS_GEMM_FP16_DEFAULT_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)     \
    using Gemm = cutlass::gemm::device::Gemm<ELEMENT_TYPE,                                                      \
                                             LAYOUT_A,                                                  \
                                             ELEMENT_TYPE,                                              \
                                             LAYOUT_B,                                                  \
                                             ELEMENT_TYPE,                                              \
                                             LAYOUT_C,                                                  \
                                             ELEMENT_ACCUMULATOR>;                                      \
    Gemm gemm_op;                                                                                       \
    Gemm::Arguments args({M, N, K},                                                                     \
                        {(ELEMENT_TYPE *) A, lda},                                                      \
                        {(ELEMENT_TYPE *) B, ldb},                                                      \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        {(ELEMENT_TYPE *) D, ldd},                                                      \
                        {alpha, beta});                                                                 \
    status = gemm_op.can_implement(args);                                                               \
    if (status == cutlass::Status::kSuccess) { \
        size_t workspace_size = Gemm::get_workspace_size(args);                                             \
        void *workspace = get_workspace(workspace_size, stream);                                            \
        gemm_op.initialize(args, workspace);                                                                \
        status = gemm_op(stream);                                                                           \
    }

#define CUTLASS_GEMM_SPLIT_K_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)          \
    using Gemm = cutlass::gemm::device::GemmSplitKParallel<ELEMENT_TYPE,                                        \
                                                            LAYOUT_A,                                          \
                                                            ELEMENT_TYPE,                                      \
                                                            LAYOUT_B,                                          \
                                                            ELEMENT_TYPE,                                      \
                                                            LAYOUT_C,                                          \
                                                            ELEMENT_ACCUMULATOR>;                              \
    Gemm gemm_op;                                                                               \
    Gemm::Arguments args({M, N, K},                                                             \
                        {(ELEMENT_TYPE *) A, lda},                                              \
                        {(ELEMENT_TYPE *) B, ldb},                                              \
                        {(ELEMENT_TYPE *) C, ldc},                                              \
                        {(ELEMENT_TYPE *) D, ldd},                                              \
                        {alpha, beta},                                                          \
                        split_k_slices);                                                        \
    status = gemm_op.can_implement(args);                                                       \
    if (status == cutlass::Status::kSuccess) { \
        size_t workspace_size = Gemm::get_workspace_size(args);                                     \
        void *workspace = get_workspace(workspace_size, stream);                                    \
        gemm_op.initialize(args, workspace);                                                        \
        status = gemm_op(stream);                                                                   \
    }

#define CUTLASS_GEMM_BATCHED_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)  \
    using Gemm = cutlass::gemm::device::GemmBatched<ELEMENT_TYPE,                                       \
                                                    LAYOUT_A,                                           \
                                                    ELEMENT_TYPE,                                       \
                                                    LAYOUT_B,                                           \
                                                    ELEMENT_TYPE,                                       \
                                                    LAYOUT_C,                                           \
                                                    ELEMENT_ACCUMULATOR>;                               \
    Gemm gemm_op;                                                                                       \
    Gemm::Arguments args({M, N, K},                                                                     \
                        {(ELEMENT_TYPE *) A, lda},                                                      \
                        batch_stride_A,                                                                 \
                        {(ELEMENT_TYPE *) B, ldb},                                                      \
                        batch_stride_B,                                                                 \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        batch_stride_C,                                                                 \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        batch_stride_C,                                                                 \
                        {alpha, beta},                                                                  \
                        batch_count);                                                                   \
    status = gemm_op.can_implement(args);                                                               \
    if (status == cutlass::Status::kSuccess) { \
        size_t workspace_size = Gemm::get_workspace_size(args);                                             \
        void *workspace = get_workspace(workspace_size, stream);                                            \
        gemm_op.initialize(args, workspace);                                                                \
        status = gemm_op(stream);                                                                           \
    }

#define CUTLASS_GEMM_BATCHED_FP16_TEMPLATE(ELEMENT_TYPE, LAYOUT_A, LAYOUT_B, LAYOUT_C, ELEMENT_ACCUMULATOR)  \
    using Gemm = cutlass::gemm::device::GemmBatched<ELEMENT_TYPE,                                       \
                                                    LAYOUT_A,                                           \
                                                    ELEMENT_TYPE,                                       \
                                                    LAYOUT_B,                                           \
                                                    ELEMENT_TYPE,                                       \
                                                    LAYOUT_C,                                           \
                                                    ELEMENT_ACCUMULATOR,    \
                                                    cutlass::arch::OpClassTensorOp,                            \
                                                    cutlass::arch::Sm80,                                       \
                                                    cutlass::gemm::GemmShape<128, 128, 16>,                    \
                                                    cutlass::gemm::GemmShape<64, 64, 16>,                      \
                                                    cutlass::gemm::GemmShape<16, 8, 8>>;                               \
    Gemm gemm_op;                                                                                       \
    Gemm::Arguments args({M, N, K},                                                                     \
                        {(ELEMENT_TYPE *) A, lda},                                                      \
                        batch_stride_A,                                                                 \
                        {(ELEMENT_TYPE *) B, ldb},                                                      \
                        batch_stride_B,                                                                 \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        batch_stride_C,                                                                 \
                        {(ELEMENT_TYPE *) C, ldc},                                                      \
                        batch_stride_C,                                                                 \
                        {alpha, beta},                                                                  \
                        batch_count);                                                                   \
    status = gemm_op.can_implement(args);                                                               \
    if (status == cutlass::Status::kSuccess) {  \
        size_t workspace_size = Gemm::get_workspace_size(args);                                             \
        void *workspace = get_workspace(workspace_size, stream);                                            \
        gemm_op.initialize(args, workspace);                                                                \
        status = gemm_op(stream);                                                                           \
    }

#define INVOKE_CUTLASS_GEMM_TEMPLATE(TEMPLATE_NAME, ELEMENT_TYPE, ElementAccumulator)                    \
    if (transA == cutlassOperation_t::CUTLASS_OP_N && transB == cutlassOperation_t::CUTLASS_OP_N) {             \
        TEMPLATE_NAME(ELEMENT_TYPE, ColumnMajor, ColumnMajor, ColumnMajor, ElementAccumulator);                 \
    } else if (transA == cutlassOperation_t::CUTLASS_OP_T && transB == cutlassOperation_t::CUTLASS_OP_N) {      \
        TEMPLATE_NAME(ELEMENT_TYPE, RowMajor, ColumnMajor, ColumnMajor, ElementAccumulator);                    \
    } else if (transA == cutlassOperation_t::CUTLASS_OP_N && transB == cutlassOperation_t::CUTLASS_OP_T) {      \
        TEMPLATE_NAME(ELEMENT_TYPE, ColumnMajor, RowMajor, ColumnMajor, ElementAccumulator);                    \
    } else if (transA == cutlassOperation_t::CUTLASS_OP_T && transB == cutlassOperation_t::CUTLASS_OP_T)  {     \
        TEMPLATE_NAME(ELEMENT_TYPE, RowMajor, RowMajor, ColumnMajor, ElementAccumulator);                       \
    } else {                                                                                                    \
        throw std::runtime_error("Not implemented.");                                                           \
    }

#define SET_SPLIT_K_SLICES(MM, NN, KK, SPLICES)         \
    if (M == MM && N == NN && K == KK) {                \
        split_k_slices = SPLICES;                       \
        use_k_split = true;                             \
    }

using RowMajor = cutlass::layout::RowMajor;
using ColumnMajor = cutlass::layout::ColumnMajor;

extern "C" {

void tally_register_cutlass()
{
    std::cout << "tally register cutlass ..." << std::endl;
}

hipError_t cutlassGemm_f32(
    cutlassOperation_t transA,
    cutlassOperation_t transB,
    int M,
    int N,
    int K,
    float alpha,
    float const *A,
    int lda,
    float const *B,
    int ldb,
    float beta,
    float *C,
    int ldc,
    float *D,
    int ldd,
    float *bias,
    void *workSpace,
    hipStream_t stream
) {

    cutlass::Status status;

    bool use_k_split = false;
    int split_k_slices = 0;

    // Hardcode the split-k slices for specific input dimensions
    SET_SPLIT_K_SLICES(1024, 60, 1024, 16);
    SET_SPLIT_K_SLICES(4096, 60, 1024, 4);
    SET_SPLIT_K_SLICES(1024, 60, 4096, 16);
    SET_SPLIT_K_SLICES(1024, 60, 96103, 128);

    if (use_k_split) {
        INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_SPLIT_K_TEMPLATE, float, float);
    } else {
        INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_TEMPLATE, float, float);
    }

    if (bias) {
        thrust::device_ptr<float> D_thrust(D);

        thrust::transform(thrust::cuda::par.on(stream),
                          D_thrust,
                          D_thrust + M * N, 
                          thrust::make_counting_iterator(0), 
                          D_thrust, 
                          AddVecBiasFunctor<float>(M, thrust::raw_pointer_cast(bias)));
    }

    CUTLASS_CHECK(status);
    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

hipError_t cutlassGemm_f16(
    cutlassOperation_t transA,
    cutlassOperation_t transB,
    int M,
    int N,
    int K,
    float alpha,
    half const *A,
    int lda,
    half const *B,
    int ldb,
    float beta,
    half *C,
    int ldc,
    half *D,
    int ldd,
    half *bias,
    hipStream_t stream
) {

    cutlass::Status status;

    // Some sizes do not work in tensor core mode because of misaligned error
    // won't fix at this point
    INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_FP16_TEMPLATE, cutlass::half_t, float);
    
    // Some sizes do not work in tensor core mode because of misaligned error
    // won't fix at this point
    if (status == cutlass::Status::kErrorMisalignedOperand) {
        INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_FP16_DEFAULT_TEMPLATE, cutlass::half_t, float);
    }

    CUTLASS_CHECK(status);
    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    if (bias) {
        thrust::device_ptr<half> D_thrust(D);

        thrust::transform(thrust::cuda::par.on(stream),
                          D_thrust,
                          D_thrust + M * N, 
                          thrust::make_counting_iterator(0), 
                          D_thrust, 
                          AddVecBiasFunctor<half>(M, thrust::raw_pointer_cast(bias)));
    }

    return hipSuccess;
}

hipError_t cutlassStridedBatchedGemm_f32(
    cutlassOperation_t transA,
    cutlassOperation_t transB,
    int M, 
    int N,
    int K,
    float alpha,
    float const *A,
    int lda,
    long long int batch_stride_A,
    float const *B,
    int ldb,
    long long int batch_stride_B,
    float *C,
    int ldc,
    long long int batch_stride_C,
    float beta,
    int batch_count,
    hipStream_t stream
) {

    cutlass::Status status;

    INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_BATCHED_TEMPLATE, float, float);

    CUTLASS_CHECK(status);
    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;

}

hipError_t cutlassStridedBatchedGemm_f16(
    cutlassOperation_t transA,
    cutlassOperation_t transB,
    int M, 
    int N,
    int K,
    float alpha,
    half const *A,
    int lda,
    long long int batch_stride_A,
    half const *B,
    int ldb,
    long long int batch_stride_B,
    half *C,
    int ldc,
    long long int batch_stride_C,
    float beta,
    int batch_count,
    hipStream_t stream
) {

    cutlass::Status status;

    INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_BATCHED_FP16_TEMPLATE, cutlass::half_t, float);

    // Some sizes do not work in tensor core mode because of misaligned error
    // won't fix at this point
    if (status == cutlass::Status::kErrorMisalignedOperand) {
        INVOKE_CUTLASS_GEMM_TEMPLATE(CUTLASS_GEMM_BATCHED_TEMPLATE, cutlass::half_t, float);
    }

    CUTLASS_CHECK(status);
    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }

    return hipSuccess;

}

}